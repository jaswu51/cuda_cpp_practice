#include <hip/hip_runtime.h>
#include <stdio.h>

// 使用extern "C"让C++函数能被Python的ctypes调用
extern "C" {
    void get_gpu_memory_info(float* free_mem, float* total_mem) {
        size_t free_memory, total_memory;
        hipError_t error = hipMemGetInfo(&free_memory, &total_memory);
        
        if(error != hipSuccess) {
            printf("CUDA错误: %s\n", hipGetErrorString(error));
            return;
        }
        
        *free_mem = free_memory/1024.0/1024.0;  // 转换为MB
        *total_mem = total_memory/1024.0/1024.0; // 转换为MB
    }
}
